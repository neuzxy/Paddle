#include "hip/hip_runtime.h"
//   Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <string>
#include "paddle/fluid/operators/fused/fused_seqpool_cvm_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

template <typename T>
__global__ void FusedSeqpoolKernel(
    T **input_values, T **seqpool_output_values, size_t **lods_values,
    const int64_t *data_lens, const int batch_size, const int embedding_size,
    const float pad_value, bool need_filter, float show_coeff, float clk_coeff,
    float threshold) {
  int bId = blockIdx.y * gridDim.x + blockIdx.x;
  int x = bId / batch_size;
  int y = bId - (x ? data_lens[x - 1] : 0);
  int start = *(lods_values[x] + y);
  int end = *(lods_values[x] + y + 1);

  for (int tid = threadIdx.x; tid < embedding_size; tid += blockDim.x) {
    if (start == end) {
      *(seqpool_output_values[x] + y * embedding_size + tid) = pad_value;
    } else {
      if (need_filter) {
        T val = static_cast<T>(0);
        for (int k = start; k < end; k++) {
          float show = *(input_values[x] + k * embedding_size);
          float click = *(input_values[x] + k * embedding_size + 1);
          if ((show - click) * show_coeff + click * clk_coeff < threshold) {
            continue;
          }
          if (tid <= 1) {  // show & click
            val += *(input_values[x] + k * embedding_size + tid);
          } else {
            val += ((int)(*(input_values[x] + k * embedding_size + tid) * 128 +
                          0.5)) /
                   128.0;
          }
        }
        *(seqpool_output_values[x] + y * embedding_size + tid) = val;
      } else {
        T val = static_cast<T>(0);
        for (int k = start; k < end; k++) {
          val += *(input_values[x] + k * embedding_size + tid);
        }
        *(seqpool_output_values[x] + y * embedding_size + tid) = val;
      }
    }
  }
}

template <typename T>
__global__ void FusedCVMKernel(T **output_values, T **seqpool_output_values,
                               const int64_t *data_lens, const int batch_size,
                               int64_t total_len, const int embedding_size,
                               bool use_cvm) {
  CUDA_KERNEL_LOOP(i, total_len * embedding_size) {
    int key = i / embedding_size;
    int offset = i % embedding_size;
    int x = key / batch_size;
    int y = key - (x ? data_lens[x - 1] : 0);
    int cvm_offset = 2;
    if (use_cvm) {
      if (offset == 0) {
        *(output_values[x] + y * embedding_size) =
            log(*(seqpool_output_values[x] + y * embedding_size) + 1);
      } else if (offset == 1) {
        *(output_values[x] + y * embedding_size + offset) =
            log(*(seqpool_output_values[x] + y * embedding_size + 1) + 1) -
            log(*(seqpool_output_values[x] + y * embedding_size) + 1);
      } else {
        *(output_values[x] + y * embedding_size + offset) =
            *(seqpool_output_values[x] + y * embedding_size + offset);
      }
    } else {
      if (offset >= cvm_offset) {
        *(output_values[x] + y * (embedding_size - cvm_offset) + offset -
          cvm_offset) =
            *(seqpool_output_values[x] + y * embedding_size + offset);
      }
    }
  }
}

template <typename T>
__global__ void FusedSeqpoolCVMGradKernel(
    T **out_grads_values, T **out_seqpool_grads_values, T **in_grads_values,
    T **cvm_values, size_t **lods_values, const int64_t *data_lens,
    const int batch_size, int64_t total_len, const int embedding_size,
    bool use_cvm) {
  CUDA_KERNEL_LOOP(i, total_len * embedding_size) {
    int key = i / embedding_size;
    int offset = i % embedding_size;
    int x = key / batch_size;
    int y = key - (x ? data_lens[x - 1] : 0);

    int cvm_offset = 2;

    if (offset < cvm_offset) {
      *(out_seqpool_grads_values[x] + y * embedding_size + offset) =
          *(cvm_values[x] + y * cvm_offset + offset);
    } else {
      if (use_cvm) {
        *(out_seqpool_grads_values[x] + y * embedding_size + offset) =
            *(out_grads_values[x] + y * embedding_size + offset);
      } else {
        *(out_seqpool_grads_values[x] + y * embedding_size + offset) =
            *(out_grads_values[x] + y * (embedding_size - cvm_offset) + offset -
              cvm_offset);
      }
    }

    int start = *(lods_values[x] + y);
    int end = *(lods_values[x] + y + 1);
    for (int k = start; k < end; k++) {
      *(in_grads_values[x] + k * embedding_size + offset) =
          *(out_seqpool_grads_values[x] + y * embedding_size + offset);
    }
  }
}

template <typename T>
void DoFusedSeqpoolCVM(const paddle::platform::Place &place,
                       T **gpu_input_values, T **gpu_output_values,
                       T **gpu_seqpool_output_values, size_t **lods_values,
                       const int64_t *data_lens, int slot_num,
                       int64_t total_len, const int embedding_size,
                       const float padding_value, bool use_cvm,
                       bool need_filter, float show_coeff, float clk_coeff,
                       float threshold) {
  auto stream = dynamic_cast<platform::CUDADeviceContext *>(
                    platform::DeviceContextPool::Instance().Get(
                        BOOST_GET_CONST(platform::CUDAPlace, place)))
                    ->stream();

  int batch_size = total_len / slot_num;
  dim3 grid(batch_size, slot_num);
  FusedSeqpoolKernel<<<grid, PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
      gpu_input_values, gpu_seqpool_output_values, lods_values, data_lens,
      batch_size, embedding_size, padding_value, need_filter, show_coeff,
      clk_coeff, threshold);

  FusedCVMKernel<<<(total_len * embedding_size + PADDLE_CUDA_NUM_THREADS - 1) /
                       PADDLE_CUDA_NUM_THREADS,
                   PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
      gpu_output_values, gpu_seqpool_output_values, data_lens, batch_size,
      total_len, embedding_size, use_cvm);
}

template <typename T>
void FusedSeqpoolCVM(const paddle::platform::Place &place,
                     const std::vector<const T *> &input_data,
                     const std::vector<T *> &output_data,
                     const std::vector<T *> &seqpool_output_data,
                     std::vector<const size_t *> lods,
                     const std::vector<int64_t> &data_lengths,
                     const int embedding_size, const float padding_value,
                     const bool use_cvm, float need_filter, float show_coeff,
                     float clk_coeff, float threshold) {
  auto data_lengths_lod = data_lengths;
  int slot_num = static_cast<int>(data_lengths.size());
  for (int i = 1; i < slot_num; i++) {
    data_lengths_lod[i] += data_lengths_lod[i - 1];
  }

  int64_t total_length = data_lengths_lod[slot_num - 1];

  auto stream = dynamic_cast<platform::CUDADeviceContext *>(
                    platform::DeviceContextPool::Instance().Get(
                        BOOST_GET_CONST(platform::CUDAPlace, place)))
                    ->stream();

  LoDTensor data_lens_tensor;
  int64_t *data_lens = reinterpret_cast<int64_t *>(
      data_lens_tensor.mutable_data<int64_t>({slot_num, 1}, place));
  hipMemcpyAsync(data_lens, data_lengths_lod.data(),
                  data_lengths_lod.size() * sizeof(int64_t),
                  hipMemcpyHostToDevice, stream);

  auto gpu_input_ptr =
      memory::AllocShared(place, input_data.size() * sizeof(T *));
  T **gpu_input_values = reinterpret_cast<T **>(gpu_input_ptr->ptr());
  hipMemcpyAsync(gpu_input_values, input_data.data(),
                  input_data.size() * sizeof(T *), hipMemcpyHostToDevice,
                  stream);

  auto gpu_output_ptr =
      memory::AllocShared(place, output_data.size() * sizeof(T *));
  T **gpu_output_values = reinterpret_cast<T **>(gpu_output_ptr->ptr());
  hipMemcpyAsync(gpu_output_values, output_data.data(),
                  output_data.size() * sizeof(T *), hipMemcpyHostToDevice,
                  stream);

  auto gpu_seqpool_output_ptr =
      memory::AllocShared(place, seqpool_output_data.size() * sizeof(T *));
  T **gpu_seqpool_output_values =
      reinterpret_cast<T **>(gpu_seqpool_output_ptr->ptr());
  hipMemcpyAsync(gpu_seqpool_output_values, seqpool_output_data.data(),
                  seqpool_output_data.size() * sizeof(T *),
                  hipMemcpyHostToDevice, stream);

  auto lods_ptr = memory::AllocShared(place, lods.size() * sizeof(size_t *));
  size_t **lods_values = reinterpret_cast<size_t **>(lods_ptr->ptr());
  hipMemcpyAsync(lods_values, lods.data(), lods.size() * sizeof(size_t *),
                  hipMemcpyHostToDevice, stream);

  DoFusedSeqpoolCVM(place, gpu_input_values, gpu_output_values,
                    gpu_seqpool_output_values, lods_values, data_lens, slot_num,
                    total_length, embedding_size, padding_value, use_cvm,
                    need_filter, show_coeff, clk_coeff, threshold);
}

template <typename T>
static void FusedSeqpoolCVMFunctor(const framework::ExecutionContext &ctx) {
  auto inputs = ctx.MultiInput<LoDTensor>("X");
  auto outputs = ctx.MultiOutput<framework::Tensor>("Out");

  const auto slot_size = inputs.size();
  std::vector<const float *> input_data(slot_size);
  std::vector<int64_t> data_lens(slot_size);
  std::vector<const size_t *> lods_data(slot_size);
  std::vector<T *> output_data(slot_size);

  std::vector<LoDTensor> seqpool_outputs(slot_size);
  std::vector<T *> seqpool_output_data(slot_size);

  auto padding_value = ctx.Attr<float>("pad_value");
  auto use_cvm = ctx.Attr<bool>("use_cvm");
  bool need_filter = ctx.Attr<bool>("need_filter");
  float show_coeff = ctx.Attr<float>("show_coeff");
  float clk_coeff = ctx.Attr<float>("clk_coeff");
  float threshold = ctx.Attr<float>("threshold");

  int embedding_size = inputs[0]->numel() / inputs[0]->dims()[0];

  for (size_t i = 0; i < slot_size; ++i) {
    const auto *input = inputs[i];
    auto dims = input->dims();

    auto lod = input->lod();
    auto lod_level = lod.size();
    int batch_size = lod[lod_level - 1].size() - 1;  // -1 to real batch size

    input_data[i] = reinterpret_cast<const T *>(input->data<T>());
    auto *output = outputs[i];
    if (use_cvm) {
      output->Resize({batch_size, embedding_size});
    } else {
      output->Resize({batch_size, embedding_size - 2});
    }
    output_data[i] =
        reinterpret_cast<T *>(output->mutable_data<T>(ctx.GetPlace()));
    data_lens[i] = lod[lod_level - 1].size() - 1;
    lods_data[i] = lod[lod_level - 1].CUDAData(ctx.GetPlace());

    seqpool_output_data[i] =
        reinterpret_cast<T *>(seqpool_outputs[i].mutable_data<T>(
            {batch_size, embedding_size}, ctx.GetPlace()));
  }

  FusedSeqpoolCVM(ctx.GetPlace(), input_data, output_data, seqpool_output_data,
                  lods_data, data_lens, embedding_size, padding_value, use_cvm,
                  need_filter, show_coeff, clk_coeff, threshold);
}

template <typename T>
void DoFusedSeqpoolCVMGrad(const paddle::platform::Place &place,
                           T **out_grads_values, T **out_seqpool_grads_values,
                           T **in_grads_values, T **gpu_cvm_values,
                           size_t **lods_values, const int64_t *slot_lens,
                           int slot_num, int64_t total_len,
                           const int embedding_size, bool use_cvm) {
  auto stream = dynamic_cast<platform::CUDADeviceContext *>(
                    platform::DeviceContextPool::Instance().Get(
                        BOOST_GET_CONST(platform::CUDAPlace, place)))
                    ->stream();
  const int batch_size = total_len / slot_num;
  FusedSeqpoolCVMGradKernel<<<(total_len * embedding_size +
                               PADDLE_CUDA_NUM_THREADS - 1) /
                                  PADDLE_CUDA_NUM_THREADS,
                              PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
      out_grads_values, out_seqpool_grads_values, in_grads_values,
      gpu_cvm_values, lods_values, slot_lens, batch_size, total_len,
      embedding_size, use_cvm);
}

template <typename T>
void FusedSeqpoolCVMGrad(const paddle::platform::Place &place,
                         const std::vector<const T *> &out_grads_data,
                         const std::vector<T *> &out_seqpool_grads_data,
                         const std::vector<T *> &in_grads_data,
                         const std::vector<const T *> &cvm_data,
                         std::vector<const size_t *> &lods,
                         const std::vector<int64_t> &data_lengths,
                         const int embedding_size, const bool use_cvm) {
  auto data_lengths_lod = data_lengths;
  int slot_num = static_cast<int>(data_lengths.size());
  for (int i = 1; i < slot_num; i++) {
    data_lengths_lod[i] += data_lengths_lod[i - 1];
  }

  int64_t total_length = data_lengths_lod[slot_num - 1];

  auto stream = dynamic_cast<platform::CUDADeviceContext *>(
                    platform::DeviceContextPool::Instance().Get(
                        BOOST_GET_CONST(platform::CUDAPlace, place)))
                    ->stream();

  LoDTensor data_lens_tensor;
  int64_t *data_lens = reinterpret_cast<int64_t *>(
      data_lens_tensor.mutable_data<int64_t>({slot_num, 1}, place));
  hipMemcpyAsync(data_lens, data_lengths_lod.data(),
                  data_lengths_lod.size() * sizeof(int64_t),
                  hipMemcpyHostToDevice, stream);

  auto gpu_out_grads_ptr =
      memory::AllocShared(place, out_grads_data.size() * sizeof(T *));
  T **gpu_out_grads_values = reinterpret_cast<T **>(gpu_out_grads_ptr->ptr());
  hipMemcpyAsync(gpu_out_grads_values, out_grads_data.data(),
                  out_grads_data.size() * sizeof(T *), hipMemcpyHostToDevice,
                  stream);

  auto gpu_out_seqpool_grads_ptr =
      memory::AllocShared(place, out_seqpool_grads_data.size() * sizeof(T *));
  T **gpu_out_seqpool_grads_values =
      reinterpret_cast<T **>(gpu_out_seqpool_grads_ptr->ptr());
  hipMemcpyAsync(gpu_out_seqpool_grads_values, out_seqpool_grads_data.data(),
                  out_seqpool_grads_data.size() * sizeof(T *),
                  hipMemcpyHostToDevice, stream);

  auto gpu_in_grads_ptr =
      memory::AllocShared(place, in_grads_data.size() * sizeof(T *));
  T **gpu_in_grads_values = reinterpret_cast<T **>(gpu_in_grads_ptr->ptr());
  hipMemcpyAsync(gpu_in_grads_values, in_grads_data.data(),
                  in_grads_data.size() * sizeof(T *), hipMemcpyHostToDevice,
                  stream);

  auto gpu_cvm_ptr = memory::AllocShared(place, cvm_data.size() * sizeof(T *));
  T **gpu_cvm_values = reinterpret_cast<T **>(gpu_cvm_ptr->ptr());
  hipMemcpyAsync(gpu_cvm_values, cvm_data.data(),
                  cvm_data.size() * sizeof(T *), hipMemcpyHostToDevice,
                  stream);

  auto lods_ptr = memory::AllocShared(place, lods.size() * sizeof(size_t *));
  size_t **lods_values = reinterpret_cast<size_t **>(lods_ptr->ptr());
  hipMemcpyAsync(lods_values, lods.data(), lods.size() * sizeof(size_t *),
                  hipMemcpyHostToDevice, stream);

  DoFusedSeqpoolCVMGrad(place, gpu_out_grads_values,
                        gpu_out_seqpool_grads_values, gpu_in_grads_values,
                        gpu_cvm_values, lods_values, data_lens, slot_num,
                        total_length, embedding_size, use_cvm);
}

template <typename T>
static void FusedSeqpoolCVMGradFunctor(const framework::ExecutionContext &ctx) {
  auto out_grads = ctx.MultiInput<LoDTensor>(framework::GradVarName("Out"));
  auto in_grads = ctx.MultiOutput<LoDTensor>(framework::GradVarName("X"));
  auto *cvm = ctx.Input<LoDTensor>("CVM");

  std::string pooltype = ctx.Attr<std::string>("pooltype");
  auto use_cvm = ctx.Attr<bool>("use_cvm");

  const auto slot_size = in_grads.size();
  std::vector<const T *> out_grads_data(slot_size);
  std::vector<T *> in_grads_data(slot_size);
  std::vector<const T *> cvm_data(slot_size);
  std::vector<const size_t *> lods_data(slot_size);
  std::vector<int64_t> data_lengths(slot_size);

  std::vector<LoDTensor> out_seqpool_grads(slot_size);
  std::vector<T *> out_seqpool_grads_data(slot_size);

  int embedding_size = in_grads[0]->numel() / in_grads[0]->dims()[0];

  for (size_t i = 0; i < slot_size; ++i) {
    auto *in_grad = in_grads[i];
    auto dims = in_grad->dims();

    auto lod = in_grad->lod();
    auto lod_level = lod.size();
    int batch_size = lod[lod_level - 1].size() - 1;  // -1 to real batch size

    auto *out_grad = out_grads[i];
    out_grads_data[i] = reinterpret_cast<const T *>(out_grad->data<T>());

    in_grads_data[i] =
        reinterpret_cast<T *>(in_grad->mutable_data<T>(ctx.GetPlace()));
    lods_data[i] = lod[lod_level - 1].CUDAData(ctx.GetPlace());
    data_lengths[i] = lod[lod_level - 1].size() - 1;
    cvm_data[i] = reinterpret_cast<const T *>(cvm->data<T>());

    out_seqpool_grads_data[i] =
        reinterpret_cast<T *>(out_seqpool_grads[i].mutable_data<T>(
            {batch_size, embedding_size}, ctx.GetPlace()));
  }

  FusedSeqpoolCVMGrad(ctx.GetPlace(), out_grads_data, out_seqpool_grads_data,
                      in_grads_data, cvm_data, lods_data, data_lengths,
                      embedding_size, use_cvm);
}

template <typename T>
class FusedSeqpoolCVMCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    FusedSeqpoolCVMFunctor<T>(ctx);
  }
};

template <typename T>
class FusedSeqpoolCVMGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    FusedSeqpoolCVMGradFunctor<T>(ctx);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(fused_seqpool_cvm,
                        ops::FusedSeqpoolCVMCUDAKernel<float>);

REGISTER_OP_CUDA_KERNEL(fused_seqpool_cvm_grad,
                        ops::FusedSeqpoolCVMGradCUDAKernel<float>);
